
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }       
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

typedef float Real;

typedef struct Array2D {
    Real* arr;        
    int rows;       
    int cols;       
} Array2D;

__global__ void kernel(const int m, const int n, Real *lval, Array2D *output)
{
    lval[threadIdx.x] = 1.0f + threadIdx.x;
    if (threadIdx.x == 0) {
        output->arr = lval;
        output->rows = m;
        output->cols = n;
    }
}

int main(void)
{
    const int m=8, n=8, mn=m*n;

    Array2D *d_output;
    Real *d_arr;
    GPUerrchk( hipMalloc((void **)&d_arr,sizeof(Real)*size_t(mn)) ); 

    GPUerrchk( hipMalloc((void **)&d_output, sizeof(Array2D)) );
    kernel<<<1,mn>>>(m,n,d_arr,d_output);
    GPUerrchk( hipPeekAtLastError() );

    // This section of code is the same as the original question
    Array2D *h_output = (Array2D*)malloc(sizeof(Array2D));
    GPUerrchk( hipMemcpy(h_output, d_output, sizeof(Array2D), hipMemcpyDeviceToHost) );
    size_t sz = size_t(h_output->rows*h_output->cols)*sizeof(Real);
    Real *h_arr = (Real*)malloc(sz);
    GPUerrchk( hipMemcpy(h_arr, h_output->arr, sz, hipMemcpyDeviceToHost) );

    for(int i=0; i<h_output->rows; i++)
        for(int j=0; j<h_output->cols; j++)
            fprintf(stdout,"(%d %d) %f\n", i, j, h_arr[j + i*h_output->rows]);

    return 0;
}
