#include "hip/hip_runtime.h"
#include<stdio.h>
#include<cutil_inline.h>
#define BLOCK_SIZE 16

texture<float,2>texVecA;
texture<float,2>texVecB;
__constant__ int ciMatSizeM;
__constant__ int ciMatSizeN;
__global__ static void AddKernel(float *d_Result)
{
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if(tidx<ciMatSizeM && tidy<ciMatSizeN)
	{
		float flValA = tex2D(texVecA,tidx,tidy);
		float flValB = tex2D(texVecB,tidx,tidy);
		d_Result[tidx * ciMatSizeN + tidy] = flValA + flValB;
	}
}

void printMatrix(float *pflMat, int iMatSizeM, int iMatSizeN)
{
	for(int idxM = 0; idxM < iMatSizeM; idxM++)
	{
		for(int idxN = 0; idxN < iMatSizeN; idxN++)
		{
			printf("%f\t",pflMat[(idxM * iMatSizeN) + idxN]);
		}
		printf("\n");
	}
	printf("\n");
}

int main()
{	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	int iMatSizeM=0,iMatSizeN=0;
	printf("Enter size of Matrix(M*N):");
	scanf("%d %d",&iMatSizeM,&iMatSizeN);

	float *h_flMat1 = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
	float *h_flMat2 = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
	float *h_flMatSum = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
	for(int j=0;j<(iMatSizeM*iMatSizeN);j++)
	{
		h_flMat1[j]=(float)rand()/(float)RAND_MAX;
		h_flMat2[j]=(float)rand()/(float)RAND_MAX;
	}

	printf("Matrix 1\n");
	printMatrix(h_flMat1, iMatSizeM, iMatSizeN);
	printf("Matrix 2\n");
	printMatrix(h_flMat2, iMatSizeM, iMatSizeN);

	float *d_flMat1 = NULL, *d_flMat2 = NULL, *d_flMatSum = NULL;
	hipMalloc(&d_flMat1, iMatSizeM * iMatSizeN * sizeof(float));
	hipMalloc(&d_flMat2, iMatSizeM * iMatSizeN * sizeof(float));
	hipMalloc(&d_flMatSum, iMatSizeM * iMatSizeN * sizeof(float));

	hipMemcpy(d_flMat1, h_flMat1, iMatSizeM * iMatSizeN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_flMat2, h_flMat2, iMatSizeM * iMatSizeN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(ciMatSizeM),&iMatSizeM,sizeof(float),0);
	hipMemcpyToSymbol(HIP_SYMBOL(ciMatSizeN),&iMatSizeN,sizeof(float),0);

	hipBindTexture2D(0, texVecA, d_flMat1, channelDesc, iMatSizeN, iMatSizeM, iMatSizeM * sizeof(float));
	hipBindTexture2D(0, texVecB, d_flMat2, channelDesc, iMatSizeN, iMatSizeM, iMatSizeM * sizeof(float));

	dim3 blocks(1,1,1);
	dim3 threadsperblock(BLOCK_SIZE,BLOCK_SIZE,1);
	blocks.x=((iMatSizeM/BLOCK_SIZE) + (((iMatSizeM)%BLOCK_SIZE)==0?0:1));
	blocks.y=((iMatSizeN/BLOCK_SIZE) + (((iMatSizeN)%BLOCK_SIZE)==0?0:1));

	AddKernel<<<blocks,threadsperblock>>>(d_flMatSum);

	hipDeviceSynchronize();
	hipMemcpy(h_flMatSum,d_flMatSum,iMatSizeM * iMatSizeN * sizeof(float), hipMemcpyDeviceToHost);
	hipUnbindTexture(texVecA);
	hipUnbindTexture(texVecB);

	hipFree(d_flMat1);
	hipFree(d_flMat2);
	hipFree(d_flMatSum);

	printf("Matrix Sum\n");
	printMatrix(h_flMatSum, iMatSizeM, iMatSizeN);
}
