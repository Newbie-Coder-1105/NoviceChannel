
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// set a 3D volume
// To compile it with nvcc execute: nvcc -O2 -o set3d set3d.cu
//define the data set size (cubic volume)
#define DATAXSIZE 100
#define DATAYSIZE 100
#define DATAZSIZE 20
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

// device function to set the 3D volume
__global__ void set(int a[][DATAYSIZE][DATAXSIZE])
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    if ((idx < (DATAXSIZE)) && (idy < (DATAYSIZE)) && (idz < (DATAZSIZE))){
      a[idz][idy][idx] = idz+idy+idx;
      }
}

int main(int argc, char *argv[])
{
    typedef int nRarray[DATAYSIZE][DATAXSIZE];
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((DATAXSIZE+BLKXSIZE-1)/BLKXSIZE), ((DATAYSIZE+BLKYSIZE-1)/BLKYSIZE), ((DATAZSIZE+BLKZSIZE-1)/BLKZSIZE));
// overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
// pointers for data set storage via malloc
    nRarray *c; // storage for result stored on host
    nRarray *d_c;  // storage for result computed on device
// allocate storage for data set
    if ((c = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
    hipMalloc((void **) &d_c, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
// compute result
    set<<<gridSize,blockSize>>>(d_c);
    cudaCheckErrors("Kernel launch failure");
// copy output data back to host

    hipMemcpy(c, d_c, ((nx*ny*nz)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
// and check for accuracy
    for (unsigned i=0; i<nz; i++)
      for (unsigned j=0; j<ny; j++)
        for (unsigned k=0; k<nx; k++)
          if (c[i][j][k] != (i+j+k)) {
            printf("Mismatch at x= %d, y= %d, z= %d  Host= %d, Device = %d\n", i, j, k, (i+j+k), c[i][j][k]);
            return 1;
            }
    printf("Results check!\n");
    free(c);
    hipFree(d_c);
    cudaCheckErrors("hipFree fail");
    return 0;
}
