
#include <hip/hip_runtime.h>
#include <cstdio>
inline void GPUassert(hipError_t code, char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }



 __global__ void doSmth(int*** a) {
  for(int i=0; i<2; i++)
   for(int j=0; j<2; j++)
    for(int k=0; k<2; k++)
     a[i][j][k]=i+j+k;
 }
 
 
 
 int main() {
  int*** h_c = (int***) malloc(2*sizeof(int**));
  for(int i=0; i<2; i++) {
   h_c[i] = (int**) malloc(2*sizeof(int*));
   for(int j=0; j<2; j++)
    GPUerrchk(hipMalloc((void**)&h_c[i][j],2*sizeof(int)));
  }
  int ***h_c1 = (int ***) malloc(2*sizeof(int **));
  for (int i=0; i<2; i++){
    GPUerrchk(hipMalloc((void***)&(h_c1[i]), 2*sizeof(int*)));
    GPUerrchk(hipMemcpy(h_c1[i], h_c[i], 2*sizeof(int*), hipMemcpyHostToDevice));
    }
  int*** d_c;
  GPUerrchk(hipMalloc((void****)&d_c,2*sizeof(int**)));
  GPUerrchk(hipMemcpy(d_c,h_c1,2*sizeof(int**),hipMemcpyHostToDevice));
  doSmth<<<1,1>>>(d_c);
  GPUerrchk(hipPeekAtLastError());
  int res[2][2][2];
  for(int i=0; i<2; i++)
   for(int j=0; j<2; j++)
    GPUerrchk(hipMemcpy(&res[i][j][0], h_c[i][j],2*sizeof(int),hipMemcpyDeviceToHost));

  for(int i=0; i<2; i++)
   for(int j=0; j<2; j++)
    for(int k=0; k<2; k++)
     printf("[%d][%d][%d]=%d\n",i,j,k,res[i][j][k]);
 }
