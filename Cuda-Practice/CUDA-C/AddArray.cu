#include "hip/hip_runtime.h"
#include "stdio.h"
#include "dev_addarray.cuh"

#define N 10

__global__ void add(int *a, int *b, int *c)
{
 int tID = blockIdx.x;
 if (tID < (N-1))
 {
 c[tID] = addarray(a, b, tID);
 }
}
int main()
{
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, N*sizeof(int));
 hipMalloc((void **) &dev_b, N*sizeof(int));
 hipMalloc((void **) &dev_c, N*sizeof(int));
 // Fill Arrays
 for (int i = 0; i < N; i++)
 {
 a[i] = i,
 b[i] = 1;
 }
 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
 add<<<N,1>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
 for (int i = 0; i < N; i++)
 {
 printf("%d + %d + %d = %d\n", a[i], b[i], a[i+1], c[i]);
 }
 return 0;
}
