#include "hip/hip_runtime.h"
/*********************************************************************/
/**
*
*    tex3d_interpl_non_normalized.cu
*
*
*    Created on : 3 December, 2018
*
*
*    Author  : Newbie-Coder-1105
*
**********************************************************************/
/*********************************************************************/

#include <iostream>
#include <fstream>
#define NX 50
#define NY 50
#define NZ 10
#define IX 100
#define IY 100
#define IZ 20
#define ifact 2

texture<float, 3> tex;

__global__ void getInterpolatedFunctionValue(float *a, float *b, float *c,  float *result) // int IX, int IY, int IZ )
{
// Calculate normalized texture coordinates
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

if ((x < IX) && (y < IY) && (z < IZ))
{

	float w = a[x] + 0.5f, v = b[y] + 0.5f, u = c[z] + 0.5f  ;
	result[z * IZ * IY + y * IZ + x ] = tex3D(tex, u, v, w );
//	printf("texObj = %f,  tu = %f, tv = %f, tw = %f\n",tex, u, v, w);
	printf("output = %f, width =%d, height = %d, depth = %d\n",result[z * IZ * IY + y * IZ + x ], x, y, z);
}
}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

int main(){

int nx=NX, ny=NY, nz=NZ;
float fff[nz][ny][nx];
float x[nx], y[ny], z[nz] ;

for(int ix=0; ix<nx; ix++)
  for(int iy=0; iy<ny; iy++)
    for(int iz=0; iz<nz; iz++){
      fff[iz][iy][ix] = (ix + iy + iz ) * 100;
    }

for(int ix=0; ix<IX; ix++)
  for(int iy=0; iy<IY; iy++)
    for(int iz=0; iz<IZ; iz++){
		x[ix] =  ix / ifact ;
		y[iz] =  iy / ifact ;
		z[iz] =  iz / ifact ;
	}
	
//*************************************************************************//
float *d_x, *d_y, *d_z, *d_result, *h_result ;


hipMalloc((void**)&d_x, IX * sizeof(float));
hipMalloc((void**)&d_y, IY * sizeof(float));
hipMalloc((void**)&d_z, IZ * sizeof(float));
hipMalloc((void**)&d_result, IX * IY * IZ * sizeof(float));


hipMemcpy( d_x, x, IX * sizeof(float), hipMemcpyHostToDevice );
hipMemcpy( d_y, y, IY * sizeof(float), hipMemcpyHostToDevice );
hipMemcpy( d_z, z, IZ * sizeof(float), hipMemcpyHostToDevice );



cudaCheckErrors("allocating an array is failed");
//*************************************************************************//
	

hipArray *d_volumeArray ;

//const hipExtent extent = make_hipExtent(nx, ny, nz);
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
hipMalloc3DArray(&d_volumeArray, &channelDesc, make_hipExtent(nz, ny, nx));
cudaCheckErrors("hipMalloc3D error");

hipMemcpy3DParms copyParams = {0};
copyParams.srcPtr   = make_hipPitchedPtr((void*)fff, sizeof(float)*nz,ny,nx);
copyParams.dstArray = d_volumeArray;
copyParams.dstPos   = make_hipPos(0,0,0);
copyParams.srcPos   = make_hipPos(0,0,0);
copyParams.extent   = make_hipExtent(nz, ny, nx);
copyParams.kind     = hipMemcpyHostToDevice;
cudaCheckErrors("copyParams3D fail");
hipMemcpy3D(&copyParams);
cudaCheckErrors("hipMemcpy3DParms fail");

tex.normalized = false;
tex.filterMode = hipFilterModeLinear;
tex.addressMode[0] = hipAddressModeClamp;
tex.addressMode[1] = hipAddressModeClamp;
tex.addressMode[2] = hipAddressModeClamp;

hipBindTextureToArray(tex, d_volumeArray, channelDesc);
cudaCheckErrors("bind fail");	

const dim3 blockSize(8, 8, 8 );
const dim3 gridSize(((IZ + blockSize.x )/blockSize.x),((IY + blockSize.y )/blockSize.y),((IX + blockSize.z)/blockSize.z));

printf("Blocksize.x = %i, blockSize.y = %i, blockSize.z = %i \n", blockSize.x, blockSize.y, blockSize.z);
printf("gridSize.x = %i, gridSize.y = %i, gridSize.z = %i \n", gridSize.x, gridSize.y, gridSize.z);

getInterpolatedFunctionValue<<<gridSize, blockSize>>>(d_x, d_y, d_z, d_result) ; // IX , IY , IZ ) ;

cudaCheckErrors("kernel fail");
hipDeviceSynchronize();	
h_result = (float*) malloc(IX * IY * IZ * sizeof(float)) ;
hipMemcpy( h_result, d_result, IX * IY * IZ * sizeof(float),hipMemcpyDeviceToHost);		
cudaCheckErrors("hipMemcpy fail");
  
hipUnbindTexture(tex);
cudaCheckErrors("unbind fail");

cudaCheckErrors("hipFree fail");
hipFreeArray(d_volumeArray);
cudaCheckErrors("free fail");

printf("success!\n");

for(int ix=0; ix<IX; ix++)
  for(int iy=0; iy<IY; iy++)
    for(int iz=0; iz<IZ; iz++){
		std::cout<<h_result[iz * IZ * IY + iy * IZ + ix ] << '\n' ;
	}



return 0;
}
