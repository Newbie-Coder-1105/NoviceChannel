#include <iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>


using namespace std;

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#define N 128
#define M 64
#define W 16

/*****************/
/* CUDA MEMCHECK */
/*****************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if ( abort )
                        exit( code );
    }
}

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/******************/
/* TEST KERNEL 3D */
/******************/
__global__ void test_kernel_3D(hipPitchedPtr devPitchedPtr)
{
    int tidx =  blockIdx.x*blockDim.x+threadIdx.x;
    int tidy =  blockIdx.y*blockDim.y+threadIdx.y;

    char* devPtr = (char*) devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * N;

    for (int w = 0; w < W; w++) {
        char* slice = devPtr + w * slicePitch;
        float* row = (float*)(slice + tidy * pitch);
        row[tidx] = row[tidx] * row[tidx];
    }
}

/********/
/* MAIN */
/********/
int main()
{
    float a[N][M][W];

    for (int i=0; i<N; i++)
        for (int j=0; j<M; j++) 
            for (int w=0; w<W; w++) {
                a[i][j][w] = 3.f;
                //printf("row %i column %i depth %i value %f \n",i,j,w,a[i][j][w]);
            }

    // --- 3D pitched allocation and host->device memcopy
    hipExtent extent = make_hipExtent(M * sizeof(float), N, W);

    hipPitchedPtr devPitchedPtr;
    gpuErrchk(hipMalloc3D(&devPitchedPtr, extent));

    hipMemcpy3DParms p = { 0 };
    p.srcPtr.ptr = a;
    p.srcPtr.pitch = M * sizeof(float);
    p.srcPtr.xsize = M;
    p.srcPtr.ysize = N;
    p.dstPtr.ptr = devPitchedPtr.ptr;
    p.dstPtr.pitch = devPitchedPtr.pitch;
    p.dstPtr.xsize = M;
    p.dstPtr.ysize = N;
    p.extent.width = M * sizeof(float);
    p.extent.height = N;
    p.extent.depth = W;
    p.kind = hipMemcpyHostToDevice;
    gpuErrchk(hipMemcpy3D(&p));

    dim3 GridSize(iDivUp(M,BLOCKSIZE_x),iDivUp(N,BLOCKSIZE_y));
    dim3 BlockSize(BLOCKSIZE_y,BLOCKSIZE_x);
    test_kernel_3D<<<GridSize,BlockSize>>>(devPitchedPtr);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    p.srcPtr.ptr = devPitchedPtr.ptr;
    p.srcPtr.pitch = devPitchedPtr.pitch;
    p.dstPtr.ptr = a;
    p.dstPtr.pitch = M * sizeof(float); 
    p.kind = hipMemcpyDeviceToHost;
    gpuErrchk(hipMemcpy3D(&p));

    for (int i=0; i<N; i++) 
        for (int j=0; j<M; j++) 
            for (int w=0; w<W; w++)
                printf("row %i column %i depth %i value %f\n",i,j,w,a[i][j][w]);

    int ch = std::cin.get();
    return 0;
}
