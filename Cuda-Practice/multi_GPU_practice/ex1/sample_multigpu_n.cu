#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NB 32
#define NT 500
#define N NB*NT



__global__ void add( double *a, double *b, double *c, const int Ns );

//===========================================
__global__ void add( double *a, double *b, double *c, const int Ns){

    int tid = threadIdx.x + blockIdx.x * blockDim.x; 

    while(tid < Ns){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }

}

//============================================
//BEGIN
//===========================================
int main( void ) {

    double *a, *b, *c;

    // allocate the memory on the CPU
    a=(double *)malloc(N*sizeof(double));
    b=(double *)malloc(N*sizeof(double));
    c=(double *)malloc(N*sizeof(double));
	
    int gpu_n;
    hipGetDeviceCount(&gpu_n);
    printf("CUDA-capable device count: %i\n", gpu_n);

	int m_size = gpu_n   ;
	double *dev_a[m_size], *dev_b[m_size], *dev_c[m_size];
	int Ns[m_size] ;

	for (int i = 0 ; i < m_size ; i++ )
	{ 
		if(i<(N%gpu_n))
		{
			Ns[i] = N/gpu_n + 1 ;
		}
		else 
		{
			Ns[i] = N/gpu_n ;
		}
		
	}
	std::cout<<"Ns is created"<<'\n' ;

	// allocate the memory on the GPUs
	for(int dev=0; dev<m_size; dev++) 
	{
		hipSetDevice(dev);
		hipMalloc( (void**)&dev_a[dev], Ns[dev] * sizeof(double) );
		hipMalloc( (void**)&dev_b[dev], Ns[dev] * sizeof(double) );
		hipMalloc( (void**)&dev_c[dev], Ns[dev] * sizeof(double) );
	}

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = (double)i;
        b[i] = (double)i*2;
    }
	std::cout<<"data created"<<'\n' ;	

 
	// copy the arrays 'a' and 'b' to the GPUs
	for(int dev=0,pos=0; dev<m_size; pos+=Ns[dev], dev++) 
	{
		hipSetDevice(dev);
		hipMemcpy( dev_a[dev], a+pos, Ns[dev] * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy( dev_b[dev], b+pos, Ns[dev] * sizeof(double), hipMemcpyHostToDevice);
		std::cout<<Ns[dev] <<'\n' ;
	}
	std::cout<<"cudamemcpy done"<<'\n' ;	


	for(int i=0;i<10000;++i) 
	{
		for(int dev=0; dev<m_size; dev++) 
		{
			hipSetDevice(dev);
			add<<<NB,NT>>>( dev_a[dev], dev_b[dev], dev_c[dev], Ns[dev] );
			hipDeviceSynchronize();
		//	std::cout<<"next Kernel started for dev "<< dev<< '\n' ;
		}
	}
	std::cout<<"kernel done"<<'\n' ;	
	
	// copy the arrays 'c' from the GPUs
	for(int dev=0,pos=0; dev<m_size; pos+=Ns[dev], dev++) 
	{
		hipSetDevice(dev);
		hipMemcpy( c+pos,dev_c[dev], Ns[dev] * sizeof(double), hipMemcpyDeviceToHost);
		std::cout<<"hipMemcpyDeviceToHost     "<<dev<<'\n' ;
	}
	std::cout<<"hipMemcpyDeviceToHost done"<<'\n' ;

    // display the results
     for (int i=0; i<N; i++) {
         printf( "%g + %g = %g\n", a[i], b[i], c[i] );
      }
    printf("\nGPU done\n");

    // free the memory allocated on the GPUS
	for(int dev=0; dev<m_size; dev++) 
	{
		hipFree( dev_a[dev] );
		hipFree( dev_b[dev] );
		hipFree( dev_c[dev] );
	}
//	for (int i = 0 ; i < N ; i++ )
//	{
//		std::cout<<c[i]<<"    "<< i <<'\n' ;
//	}

	std::cout<<"cudafree done"<<'\n' ;

    // free the memory allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}
