#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NB 32
#define NT 500
#define N NB*NT

__global__ void add( double *a, double *b, double *c, const int Ns );

//===========================================
__global__ void add( double *a, double *b, double *c, const int Ns){

    int tid = threadIdx.x + blockIdx.x * blockDim.x; 

    while(tid < Ns){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }

}

//============================================
//BEGIN
//===========================================
int main( void ) {

    double *a, *b, *c;

    // allocate the memory on the CPU
    a=(double *)malloc(N*sizeof(double));
    b=(double *)malloc(N*sizeof(double));
    c=(double *)malloc(N*sizeof(double));


	double *dev_a[2], *dev_b[2], *dev_c[2];
	const int Ns[2] = {N/2, N-(N/2)};

	// allocate the memory on the GPUs
	for(int dev=0; dev<2; dev++) 
	{
		hipSetDevice(dev);
		hipMalloc( (void**)&dev_a[dev], Ns[dev] * sizeof(double) );
		hipMalloc( (void**)&dev_b[dev], Ns[dev] * sizeof(double) );
		hipMalloc( (void**)&dev_c[dev], Ns[dev] * sizeof(double) );
	}

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = (double)i;
        b[i] = (double)i*2;
    }

 
	// copy the arrays 'a' and 'b' to the GPUs
	for(int dev=0,pos=0; dev<2; pos+=Ns[dev], dev++) 
	{
		hipSetDevice(dev);
		hipMemcpy( dev_a[dev], a+pos, Ns[dev] * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy( dev_b[dev], b+pos, Ns[dev] * sizeof(double), hipMemcpyHostToDevice);
	}


	for(int i=0;i<10000;++i) 
	{
		for(int dev=0; dev<2; dev++) 
		{
			hipSetDevice(dev);
			add<<<NB,NT>>>( dev_a[dev], dev_b[dev], dev_c[dev], Ns[dev] );
			std::cout<<"next Kernel started for dev "<< dev<< '\n' ;
		}
	}
	
	// copy the arrays 'c' from the GPUs
	for(int dev=0,pos=0; dev<2; pos+=Ns[dev], dev++) 
	{
		hipSetDevice(dev);
		hipMemcpy( c+pos,dev_c[dev], Ns[dev] * sizeof(double), hipMemcpyDeviceToHost);
	}

    // display the results
    // for (int i=0; i<N; i++) {
    //      printf( "%g + %g = %g\n", a[i], b[i], c[i] );
    //  }
    printf("\nGPU done\n");

    // free the memory allocated on the GPUS
	for(int dev=0; dev<2; dev++) 
	{
		hipFree( dev_a[dev] );
		hipFree( dev_b[dev] );
		hipFree( dev_c[dev] );
	}
//	for (int i = 0 ; i < N ; i++ )
//	{
//		std::cout<<c[i]<<"    "<< i <<'\n' ;
//	}

    // free the memory allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}
