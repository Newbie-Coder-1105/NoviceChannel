#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

#include "LonGrid.h"
using namespace std;

__global__ void lonkernel(float* d_new_lon, int nlons, float del_lon)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < nlons)
	{
		float a_lon = 0.0;
		d_new_lon[x] = a_lon + (x) * del_lon ;
	}
}



float NewLonGrid::new_lon_grid(float *h_new_lon, int &NLON, float del_lon)
{
//	const int NLON = 144;
	float *d_new_lon;
//	float del_lon = 1.25;
	NLON = ( 360 + 0 )/del_lon;
	size_t nxx_size = NLON * sizeof( float ); 
    h_new_lon = (float *)malloc( nxx_size );

	hipMalloc( (void **)&d_new_lon, nxx_size ); 

	int blocknum;
	int blocksize;

	blocksize=8;
	blocknum=(int)ceil((float)NLON/8);
	
	// execute device kernel

	lonkernel<<<blocknum,blocksize>>>(d_new_lon, NLON, del_lon);
	
	hipMemcpy(h_new_lon,d_new_lon,nxx_size,hipMemcpyDeviceToHost);

	hipFree(d_new_lon);
	
	for (int i =0 ; i< NLON; i++)
	{
		cout << "*****lonpos["<<i<<"] = "<< h_new_lon[i]<<endl;
	}
	cout<<"NLON ="<<NLON<<endl;
	hipDeviceSynchronize();
	return 0;
	
}